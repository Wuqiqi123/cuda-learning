
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void gpu(){
    // blockIdx.x * blockDim.x + threadIdx.x;
    printf("hello gpu, blockIdx.x: %d, blockDim.x: %d, threadIdx.x: %d  \n", blockIdx.x, blockDim.x, threadIdx.x);
}

int main() {
    std::cout << "Hello, World!" << std::endl;

    gpu<<<2,3>>>();  // gpu配置 《block数 线程数》 不用for循环 O(n)的算法直接变成O(1)
    hipDeviceSynchronize();

    return 0;
}
